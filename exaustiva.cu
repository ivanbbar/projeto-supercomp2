#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/copy.h>

using namespace std;

struct Movie
{
  int id;
  int start;
  int end;
  int category;
};

void fillTimeSlots(int &availableTimeSlots, int start, int end)
{
  for (int i = start; i < end; i++)
  {
    availableTimeSlots |= (1 << i);
  }
}

class ExhaustiveSearchGPU
{
private:
  int movieCount;
  int categoryCount;
  int *categoryAvailability;
  int *movieSchedules;
  int *movieCategories;
  int *maxCount;

public:
  ExhaustiveSearchGPU(int movieCount_, int categoryCount_, int *categoryAvailability_, int *movieSchedules_, int *movieCategories_, int *maxCount_)
      : movieCount(movieCount_),
        categoryCount(categoryCount_),
        categoryAvailability(categoryAvailability_),
        movieSchedules(movieSchedules_),
        movieCategories(movieCategories_),
        maxCount(maxCount_) {}

  __device__ void operator()(const int &config)
  {
    int availableTimeSlots = 0;
    int viewedCategories[16];

    for (int i = 0; i < categoryCount; i++)
    {
      viewedCategories[i] = categoryAvailability[i];
    }

    int selectedMoviesCount = 0;
    for (int i = 0; i < movieCount; i++)
    {
      if (config & (1 << i))
      {
        if (viewedCategories[movieCategories[i] - 1] > 0)
        {
          int currentSchedule = availableTimeSlots & movieSchedules[i];
          if (currentSchedule != 0)
            return;

          viewedCategories[movieCategories[i] - 1]--;
          availableTimeSlots |= movieSchedules[i];
          selectedMoviesCount++;
        }
      }
    }

    atomicMax(maxCount, selectedMoviesCount);
  }
};

void inputCategoryAvailability(thrust::host_vector<int> &categoryAvailability)
{
  for (int i = 0; i < categoryAvailability.size(); i++)
  {
    cin >> categoryAvailability[i];
  }
}

void inputMovies(vector<Movie> &moviesVector, int movieCount)
{
  for (int i = 0; i < movieCount; i++)
  {
    int start, end, category;
    cin >> start >> end >> category;

    if (start > end)
    {
      if (end == 0)
      {
        end = 24;
      }
      else
      {
        continue;
      }
    }

    Movie movie;
    movie.id = i + 1;
    movie.start = start;
    movie.end = end;
    movie.category = category;

    moviesVector.push_back(movie);
  }
}

void populateMoviesAndSchedules(thrust::host_vector<int> &movieSchedulesCPU, thrust::host_vector<int> &movieCategories, const vector<Movie> &moviesVector)
{
  for (int i = 0; i < moviesVector.size(); i++)
  {
    movieSchedulesCPU[i] = 0;
    fillTimeSlots(movieSchedulesCPU[i], moviesVector[i].start, moviesVector[i].end);
    movieCategories[i] = moviesVector[i].category;
  }
}

void printSelectedMovies(const vector<Movie> &moviesVector, const thrust::host_vector<int> &configVectorCPU, int maxCount)
{
  int maxConfig = -1;
  for (int i = 0; i < configVectorCPU.size(); i++)
  {
    if (configVectorCPU[i] == maxCount)
    {
      maxConfig = i;
      break;
    }
  }

  for (int i = 0; i < moviesVector.size(); i++)
  {
    if (maxConfig & (1 << i))
    {
      cout << moviesVector[i].id << " " << moviesVector[i].category << endl;
    }
  }
}

int main()
{
  int movieCount, categoryCount;
  cin >> movieCount >> categoryCount;

  vector<Movie> moviesVector;
  thrust::host_vector<int> categoryAvailability(categoryCount);
  inputCategoryAvailability(categoryAvailability);

  inputMovies(moviesVector, movieCount);
  int actualMovieCount = moviesVector.size();

  thrust::host_vector<int> movieCategories(actualMovieCount);
  thrust::host_vector<int> movieSchedulesCPU(actualMovieCount);
  populateMoviesAndSchedules(movieSchedulesCPU, movieCategories, moviesVector);

  thrust::device_vector<int> gpuPossibilitiesVector(pow(2, actualMovieCount));
  thrust::sequence(gpuPossibilitiesVector.begin(), gpuPossibilitiesVector.end());

  thrust::device_vector<int> categoryAvailabilityGPU(categoryAvailability);
  thrust::device_vector<int> movieSchedulesGPU(movieSchedulesCPU);
  thrust::device_vector<int> movieCategoriesGPU(movieCategories);
  thrust::device_vector<int> gpuMaxCount(1, 0);

  ExhaustiveSearchGPU searchFunctor(actualMovieCount, categoryCount, raw_pointer_cast(categoryAvailabilityGPU.data()),
                                    raw_pointer_cast(movieSchedulesGPU.data()),
                                    raw_pointer_cast(movieCategoriesGPU.data()), raw_pointer_cast(gpuMaxCount.data()));

  thrust::for_each(gpuPossibilitiesVector.begin(), gpuPossibilitiesVector.end(), searchFunctor);

  thrust::host_vector<int> finalConfigVectorCPU = gpuPossibilitiesVector;
  int maxCount = *thrust::max_element(finalConfigVectorCPU.begin(), finalConfigVectorCPU.end());

  cout << maxCount << endl;
  printSelectedMovies(moviesVector, finalConfigVectorCPU, maxCount);

  return 0;
}
